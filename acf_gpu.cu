#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <chrono>

const int NUM_BINS = 10;
const double MAX_DISTANCE = 10.0;  // degrees
const double DEGREE_TO_RAD = 3.14159265358979323846 / 180.0;

__device__ double angularDistance(double ra1_deg, double dec1_deg, double ra2_deg, double dec2_deg) {
    double ra1 = ra1_deg * DEGREE_TO_RAD;
    double dec1 = dec1_deg * DEGREE_TO_RAD;
    double ra2 = ra2_deg * DEGREE_TO_RAD;
    double dec2 = dec2_deg * DEGREE_TO_RAD;

    double cos_angle = sin(dec1) * sin(dec2) + cos(dec1) * cos(dec2) * cos(ra1 - ra2);
    cos_angle = fmin(fmax(cos_angle, -1.0), 1.0); // Clamp to [-1, 1]

    double angle_rad = acos(cos_angle);
    return angle_rad / DEGREE_TO_RAD;  // convert back to degrees
}

__global__ void computeACF(const double* ra, const double* dec, int num_points, int* histogram, double bin_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_points) return;

    double ra1 = ra[i];
    double dec1 = dec[i];

    for (int j = i + 1; j < num_points; ++j) {
        double dist = angularDistance(ra1, dec1, ra[j], dec[j]);
        if (dist < MAX_DISTANCE) {
            int bin_idx = static_cast<int>(dist / bin_size);
            if (bin_idx < NUM_BINS) {
                atomicAdd(&histogram[bin_idx], 1);
            }
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " datafile.txt\n";
        return 1;
    }

    std::ifstream infile(argv[1]);
    if (!infile) {
        std::cerr << "Error opening file " << argv[1] << "\n";
        return 1;
    }

    std::vector<double> ra_vec, dec_vec;
    double ra, dec;
    while (infile >> ra >> dec) {
        ra_vec.push_back(ra);
        dec_vec.push_back(dec);
    }
    infile.close();

    int num_points = ra_vec.size();
    std::cout << "Calculating ACF for " << num_points << " points\n";

    double bin_size = MAX_DISTANCE / NUM_BINS;

    // memory allocation
    int* d_histogram;
    double* d_ra;
    double* d_dec;

    hipMalloc(&d_ra, num_points * sizeof(double));
    hipMalloc(&d_dec, num_points * sizeof(double));
    hipMalloc(&d_histogram, NUM_BINS * sizeof(int));
    hipMemset(d_histogram, 0, NUM_BINS * sizeof(int));

    // Copy data to device
    hipMemcpy(d_ra, ra_vec.data(), num_points * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_dec, dec_vec.data(), num_points * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock;

    // Timing Start
    auto start = std::chrono::high_resolution_clock::now();

    computeACF<<<blocksPerGrid, threadsPerBlock>>>(d_ra, d_dec, num_points, d_histogram, bin_size);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    // Copy histogram back to host
    std::vector<int> histogram(NUM_BINS);
    hipMemcpy(histogram.data(), d_histogram, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);

    // Output to console
    std::cout << "Angular Correlation Function Histogram:\n";
    for (int i = 0; i < NUM_BINS; ++i) {
        std::cout << i * bin_size << "-" << (i + 1) * bin_size << " deg: " << histogram[i] << "\n";
    }
    std::cout << "Time taken (GPU): " << elapsed.count() << " seconds\n";

    // Output to file in append mode
    std::ofstream outfile("acf_results_gpu.txt", std::ios::app);
    outfile << "=== Results for file: " << argv[1] << " ===\n";
    outfile << "Angular Correlation Function Histogram:\n";
    for (int i = 0; i < NUM_BINS; ++i) {
        outfile << i * bin_size << "-" << (i + 1) * bin_size << " deg: " << histogram[i] << "\n";
    }
    outfile << "Time taken (GPU): " << elapsed.count() << " seconds\n\n";
    outfile.close();

    // Cleanup
    hipFree(d_ra);
    hipFree(d_dec);
    hipFree(d_histogram);

    return 0;
}
